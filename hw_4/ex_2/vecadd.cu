
#include <hip/hip_runtime.h>
#include <iostream>
#include <random>
#include <chrono>
#include <string>

#define DataType double
#define TPB 32
#define VEC_RANGE 1e10

__global__ void vecAdd(DataType *in1, DataType *in2, DataType *out, int len) {
  //@@ Insert code to implement vector addition here
  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i >= len) return;
  out[i] = in1[i] + in2[i];
}

//@@ Insert code to implement timer
class Timer{
  std::chrono::time_point<std::chrono::high_resolution_clock> start;

public:
  Timer(){
    start = std::chrono::high_resolution_clock::now();
  }

  double get(){
    auto stop = std::chrono::high_resolution_clock::now();
    return std::chrono::duration<double>(stop - start).count();
  }
};

int main(int argc, char **argv) {
  hipFree(0);
  
  int inputLength;
  DataType *hostInput1;
  DataType *hostInput2;
  DataType *hostOutput;
  DataType *resultRef;
  DataType *deviceInput1;
  DataType *deviceInput2;
  DataType *deviceOutput;

  //@@ Insert code below to read in inputLength from args
  if(argc > 1) inputLength = std::stoi(argv[1]);
  else { 
    std::cout << "Input length was not provided";
    return;
  }

  printf("The input length is %d\n", inputLength);
  
  //@@ Insert code below to allocate Host memory for input and output
  hipHostAlloc(&hostInput1, sizeof(DataType) * inputLength, hipHostMallocDefault);
  hipHostAlloc(&hostInput2, sizeof(DataType) * inputLength, hipHostMallocDefault);
  hipHostAlloc(&hostOutput, sizeof(DataType) * inputLength, hipHostMallocDefault);
  
  //@@ Insert code below to initialize hostInput1 and hostInput2 to random numbers, and create reference result in CPU
  std::random_device rd;
  std::mt19937 gen(rd());
  std::uniform_real_distribution<DataType> dis(-VEC_RANGE, VEC_RANGE);
  for(int i = 0; i < inputLength; i++){
    hostInput1[i] = dis(gen);
    hostInput2[i] = dis(gen);
  }

  // calc reference result
  Timer cpuTimer;
  resultRef = (DataType*) malloc(sizeof(DataType) * inputLength);
  for(int i = 0; i < inputLength; i++) 
    resultRef[i] = hostInput1[i] + hostInput2[i];
  printf("CPU time: %fs\n", cpuTimer.get());

  Timer gpuTimer;
  //@@ Insert code below to allocate GPU memory here
  Timer gpuMallocTimer;
  hipMalloc(&deviceInput1, sizeof(DataType) * inputLength);
  hipMalloc(&deviceInput2, sizeof(DataType) * inputLength);
  hipMalloc(&deviceOutput, sizeof(DataType) * inputLength);
  printf("GPU malloc time: %fs\n", gpuMallocTimer.get());

  //@@ Insert code to below to Copy memory to the GPU here
  Timer h2dTimer;
  hipMemcpy(deviceInput1, hostInput1, sizeof(DataType) * inputLength, hipMemcpyHostToDevice);
  hipMemcpy(deviceInput2, hostInput2, sizeof(DataType) * inputLength, hipMemcpyHostToDevice);
  printf("Host to device copy time: %fs\n", h2dTimer.get());

  //@@ Initialize the 1D grid and block dimensions here
  dim3 blockSize(TPB);
  dim3 gridSize((inputLength + TPB - 1)/TPB);

  //@@ Launch the GPU Kernel here
  Timer kernelTimer;
  vecAdd<<<gridSize,blockSize>>>(deviceInput1, deviceInput2, deviceOutput, inputLength);
  hipDeviceSynchronize();
  printf("Kernel time: %fs\n", kernelTimer.get());

  //@@ Copy the GPU memory back to the CPU here
  Timer d2hTimer;
  hipMemcpy(hostOutput, deviceOutput, sizeof(DataType) * inputLength, hipMemcpyDeviceToHost);
  printf("Device to host copy time: %fs\n", d2hTimer.get());
  
  printf("GPU time: %fs\n", gpuTimer.get());

  //@@ Insert code below to compare the output with the reference
  printf("VERIFYING\n");
  bool isCorrect = true;
  for(int i = 0; i < inputLength; i++) {
    if(resultRef[i] != hostOutput[i]) { 
      isCorrect = false;
      break;
    }
  }

  if (isCorrect)
    printf("Correct\n");
  else
    printf("CPU and GPU result differ\n");

  //@@ Free the GPU memory here
  hipFree(deviceInput1);
  hipFree(deviceInput2);
  hipFree(deviceOutput);

  //@@ Free the CPU memory here
  hipHostFree(hostInput1);
  hipHostFree(hostInput2);
  hipHostFree(hostOutput);

  return 0;
}
