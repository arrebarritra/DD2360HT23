#include "hip/hip_runtime.h"
#include "EMfield.h"

/** allocate electric and magnetic field */
void field_allocate(struct grid* grd, struct EMfield* field)
{
    // E on nodes
    field->Ex = newArr3<FPfield>(&field->Ex_flat, grd->nxn, grd->nyn, grd->nzn);
    field->Ey = newArr3<FPfield>(&field->Ey_flat, grd->nxn, grd->nyn, grd->nzn);
    field->Ez = newArr3<FPfield>(&field->Ez_flat, grd->nxn, grd->nyn, grd->nzn);
    // B on nodes
    field->Bxn = newArr3<FPfield>(&field->Bxn_flat, grd->nxn, grd->nyn, grd->nzn);
    field->Byn = newArr3<FPfield>(&field->Byn_flat, grd->nxn, grd->nyn, grd->nzn);
    field->Bzn = newArr3<FPfield>(&field->Bzn_flat, grd->nxn, grd->nyn, grd->nzn);
}

/** deallocate electric and magnetic field */
void field_deallocate(struct grid* grd, struct EMfield* field)
{
    // E deallocate 3D arrays
    delArr3(field->Ex, grd->nxn, grd->nyn);
    delArr3(field->Ey, grd->nxn, grd->nyn);
    delArr3(field->Ez, grd->nxn, grd->nyn);

    // B deallocate 3D arrays
    delArr3(field->Bxn, grd->nxn, grd->nyn);
    delArr3(field->Byn, grd->nxn, grd->nyn);
    delArr3(field->Bzn, grd->nxn, grd->nyn);
}

#ifdef GPU

/** allocate electric and magnetic field */
void field_allocate_device(struct grid* grd, struct EMfield* d_field) {
    hipMalloc(&d_field, sizeof(EMfield));

    // E on nodes
    newArr3<FPfield><<<1,1>>>(&d_field->Ex, &d_field->Ex_flat, grd->nxn, grd->nyn, grd->nzn);
    newArr3<FPfield><<<1,1>>>(&d_field->Ey, &d_field->Ey_flat, grd->nxn, grd->nyn, grd->nzn);
    newArr3<FPfield><<<1,1>>>(&d_field->Ez, &d_field->Ez_flat, grd->nxn, grd->nyn, grd->nzn);
    
    // B on nodes
    newArr3<FPfield><<<1,1>>>(&d_field->Bxn, &d_field->Bxn_flat, grd->nxn, grd->nyn, grd->nzn);
    newArr3<FPfield><<<1,1>>>(&d_field->Byn, &d_field->Byn_flat, grd->nxn, grd->nyn, grd->nzn);
    newArr3<FPfield><<<1,1>>>(&d_field->Bzn, &d_field->Bzn_flat, grd->nxn, grd->nyn, grd->nzn);
}

__global__ void field_deallocate_kernel(struct grid* grd, struct EMfield* field) {
    
    // E deallocate 3D arrays
    delArr3(field->Ex, grd->nxn, grd->nyn);
    delArr3(field->Ey, grd->nxn, grd->nyn);
    delArr3(field->Ez, grd->nxn, grd->nyn);

    // B deallocate 3D arrays
    delArr3(field->Bxn, grd->nxn, grd->nyn);
    delArr3(field->Byn, grd->nxn, grd->nyn);
    delArr3(field->Bzn, grd->nxn, grd->nyn);

}

/** deallocate electric and magnetic field */
void field_deallocate_device(struct grid* d_grd, struct EMfield* d_field) {    
    field_deallocate_kernel<<<1,1>>>(d_grd, d_field);
}

/** synchronize */
void field_synchronize_host(struct grid* grd, struct EMfield* h_field, struct EMfield* d_field) {
    hipMemcpy(h_field, d_field, sizeof(EMfield), hipMemcpyDeviceToHost);

    FPfield*** d_Ex_flat, *** d_Ey_flat, *** d_Ez_flat;
    hipMemcpy(&d_Ex_flat, &d_field->Ex_flat, sizeof(FPfield), hipMemcpyDeviceToHost);
    hipMemcpy(&d_Ey_flat, &d_field->Ey_flat, sizeof(FPfield), hipMemcpyDeviceToHost);
    hipMemcpy(&d_Ez_flat, &d_field->Ez_flat, sizeof(FPfield), hipMemcpyDeviceToHost);

    hipMemcpy(h_field->Ex_flat, d_Ex_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyDeviceToHost);
    hipMemcpy(h_field->Ey_flat, d_Ey_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyDeviceToHost);
    hipMemcpy(h_field->Ez_flat, d_Ez_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyDeviceToHost);

    FPfield*** d_Bxn_flat, *** d_Byn_flat, *** d_Bzn_flat;
    hipMemcpy(&d_Bxn_flat, &(d_field->Bxn_flat), sizeof(FPfield), hipMemcpyDeviceToHost);
    hipMemcpy(&d_Byn_flat, &(d_field->Byn_flat), sizeof(FPfield), hipMemcpyDeviceToHost);
    hipMemcpy(&d_Bzn_flat, &(d_field->Bzn_flat), sizeof(FPfield), hipMemcpyDeviceToHost);

    hipMemcpy(h_field->Bxn_flat, d_Bxn_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyDeviceToHost);
    hipMemcpy(h_field->Byn_flat, d_Byn_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyDeviceToHost);
    hipMemcpy(h_field->Bzn_flat, d_Bzn_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyDeviceToHost);
}

void field_synchronize_device(struct grid* grd, struct EMfield* h_field, struct EMfield* d_field) {
    hipMemcpy(d_field, h_field, sizeof(EMfield), hipMemcpyDeviceToHost);

    FPfield*** d_Ex_flat, *** d_Ey_flat, *** d_Ez_flat;
    hipMemcpy(&d_Ex_flat, &(d_field->Ex_flat), sizeof(FPfield), hipMemcpyDeviceToHost);
    hipMemcpy(&d_Ey_flat, &(d_field->Ey_flat), sizeof(FPfield), hipMemcpyDeviceToHost);
    hipMemcpy(&d_Ez_flat, &(d_field->Ez_flat), sizeof(FPfield), hipMemcpyDeviceToHost);

    hipMemcpy(d_Ex_flat, h_field->Ex_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyHostToDevice);
    hipMemcpy(d_Ey_flat, h_field->Ey_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyHostToDevice);
    hipMemcpy(d_Ez_flat, h_field->Ez_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyHostToDevice);

    FPfield*** d_Bxn_flat, *** d_Byn_flat, *** d_Bzn_flat;
    hipMemcpy(&d_Bxn_flat, &(d_field->Bxn_flat), sizeof(FPfield), hipMemcpyDeviceToHost);
    hipMemcpy(&d_Byn_flat, &(d_field->Byn_flat), sizeof(FPfield), hipMemcpyDeviceToHost);
    hipMemcpy(&d_Bzn_flat, &(d_field->Bzn_flat), sizeof(FPfield), hipMemcpyDeviceToHost);

    hipMemcpy(d_Bxn_flat, h_field->Bxn_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyHostToDevice);
    hipMemcpy(d_Byn_flat, h_field->Byn_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyHostToDevice);
    hipMemcpy(d_Bzn_flat, h_field->Bzn_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyHostToDevice);
}

#endif // GPU
