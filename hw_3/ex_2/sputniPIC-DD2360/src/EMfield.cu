#include "EMfield.h"

/** allocate electric and magnetic field */
void field_allocate(struct grid* grd, struct EMfield* field)
{
    // E on nodes
    field->Ex = newArr3<FPfield>(&field->Ex_flat, grd->nxn, grd->nyn, grd->nzn);
    field->Ey = newArr3<FPfield>(&field->Ey_flat, grd->nxn, grd->nyn, grd->nzn);
    field->Ez = newArr3<FPfield>(&field->Ez_flat, grd->nxn, grd->nyn, grd->nzn);
    // B on nodes
    field->Bxn = newArr3<FPfield>(&field->Bxn_flat, grd->nxn, grd->nyn, grd->nzn);
    field->Byn = newArr3<FPfield>(&field->Byn_flat, grd->nxn, grd->nyn, grd->nzn);
    field->Bzn = newArr3<FPfield>(&field->Bzn_flat, grd->nxn, grd->nyn, grd->nzn);
}

/** deallocate electric and magnetic field */
void field_deallocate(struct grid* grd, struct EMfield* field)
{
    // E deallocate 3D arrays
    delArr3(field->Ex, grd->nxn, grd->nyn);
    delArr3(field->Ey, grd->nxn, grd->nyn);
    delArr3(field->Ez, grd->nxn, grd->nyn);

    // B deallocate 3D arrays
    delArr3(field->Bxn, grd->nxn, grd->nyn);
    delArr3(field->Byn, grd->nxn, grd->nyn);
    delArr3(field->Bzn, grd->nxn, grd->nyn);
}

#ifdef GPU

/** allocate electric and magnetic field */
void field_allocate_device(struct grid* grd, struct EMfield** p_d_field) {
    hipMalloc(p_d_field, sizeof(EMfield));
    EMfield* d_field = *p_d_field;

    // E on nodes
    FPfield ***d_Ex, ***d_Ey, ***d_Ez;
    d_Ex = newArr3_device<FPfield>(&d_field->Ex_flat, grd->nxn, grd->nyn, grd->nzn);
    d_Ey = newArr3_device<FPfield>(&d_field->Ey_flat, grd->nxn, grd->nyn, grd->nzn);
    d_Ez = newArr3_device<FPfield>(&d_field->Ez_flat, grd->nxn, grd->nyn, grd->nzn);

    hipMemcpy(&d_field->Ex, &d_Ex, sizeof(FPfield***), hipMemcpyHostToDevice);
    hipMemcpy(&d_field->Ey, &d_Ey, sizeof(FPfield***), hipMemcpyHostToDevice);
    hipMemcpy(&d_field->Ez, &d_Ez, sizeof(FPfield***), hipMemcpyHostToDevice);

    // B on nodes
    FPfield ***d_Bxn, ***d_Byn, ***d_Bzn;
    d_Bxn = newArr3_device<FPfield>(&d_field->Bxn, &d_field->Bxn_flat, grd->nxn, grd->nyn, grd->nzn);
    d_Byn = newArr3_device<FPfield>(&d_field->Byn, &d_field->Byn_flat, grd->nxn, grd->nyn, grd->nzn);
    d_Bzn = newArr3_device<FPfield>(&d_field->Bzn, &d_field->Bzn_flat, grd->nxn, grd->nyn, grd->nzn);

    hipMemcpy(&d_field->Bxn, &d_Bxn, sizeof(FPfield***), hipMemcpyHostToDevice);
    hipMemcpy(&d_field->Byn, &d_Byn, sizeof(FPfield***), hipMemcpyHostToDevice);
    hipMemcpy(&d_field->Bzn, &d_Bzn, sizeof(FPfield***), hipMemcpyHostToDevice);
}

/** deallocate electric and magnetic field */
void field_deallocate_device(struct grid* d_grd, struct EMfield* d_field) {    
    // E deallocate 3D arrays
    FPfield ***d_Ex, ***d_Ey, ***d_Ez;
    hipMemcpy(&d_Ex, &d_field->Ex, sizeof(FPfield***), hipMemcpyDeviceToHost);
    hipMemcpy(&d_Ey, &d_field->Ey, sizeof(FPfield***), hipMemcpyDeviceToHost);
    hipMemcpy(&d_Ez, &d_field->Ez, sizeof(FPfield***), hipMemcpyDeviceToHost);

    delArr3_device(d_Ex, grd->nxn, grd->nyn);
    delArr3_device(d_Ey, grd->nxn, grd->nyn);
    delArr3_device(d_Ez, grd->nxn, grd->nyn);

    // B deallocate 3D arrays
    FPfield*** d_Bxn, *** d_Byn, *** d_Bzn;
    hipMemcpy(&d_Bxn, &d_field->Bxn, sizeof(FPfield***), hipMemcpyDeviceToHost);
    hipMemcpy(&d_Byn, &d_field->Byn, sizeof(FPfield***), hipMemcpyDeviceToHost);
    hipMemcpy(&d_Bzn, &d_field->Bzn, sizeof(FPfield***), hipMemcpyDeviceToHost);

    delArr3_device(d_Bxn, grd->nxn, grd->nyn);
    delArr3_device(d_Byn, grd->nxn, grd->nyn);
    delArr3_device(d_Bzn, grd->nxn, grd->nyn);
}

/** synchronize */
void field_synchronize_host(struct grid* grd, struct EMfield* h_field, struct EMfield* d_field) {
    hipMemcpy(h_field, d_field, sizeof(EMfield), hipMemcpyDeviceToHost);

    FPfield* d_Ex_flat, * d_Ey_flat, * d_Ez_flat;
    hipMemcpy(&d_Ex_flat, &d_field->Ex_flat, sizeof(FPfield*), hipMemcpyDeviceToHost);
    hipMemcpy(&d_Ey_flat, &d_field->Ey_flat, sizeof(FPfield*), hipMemcpyDeviceToHost);
    hipMemcpy(&d_Ez_flat, &d_field->Ez_flat, sizeof(FPfield*), hipMemcpyDeviceToHost);

    hipMemcpy(h_field->Ex_flat, d_Ex_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyDeviceToHost);
    hipMemcpy(h_field->Ey_flat, d_Ey_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyDeviceToHost);
    hipMemcpy(h_field->Ez_flat, d_Ez_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyDeviceToHost);

    FPfield* d_Bxn_flat, * d_Byn_flat, * d_Bzn_flat;
    hipMemcpy(&d_Bxn_flat, &d_field->Bxn_flat, sizeof(FPfield*), hipMemcpyDeviceToHost);
    hipMemcpy(&d_Byn_flat, &d_field->Byn_flat, sizeof(FPfield*), hipMemcpyDeviceToHost);
    hipMemcpy(&d_Bzn_flat, &d_field->Bzn_flat, sizeof(FPfield*), hipMemcpyDeviceToHost);

    hipMemcpy(h_field->Bxn_flat, d_Bxn_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyDeviceToHost);
    hipMemcpy(h_field->Byn_flat, d_Byn_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyDeviceToHost);
    hipMemcpy(h_field->Bzn_flat, d_Bzn_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyDeviceToHost);
}

void field_synchronize_device(struct grid* grd, struct EMfield* h_field, struct EMfield* d_field) {
    hipMemcpy(d_field, h_field, sizeof(EMfield), hipMemcpyDeviceToHost);

    FPfield* d_Ex_flat, * d_Ey_flat, * d_Ez_flat;
    hipMemcpy(&d_Ex_flat, &d_field->Ex_flat, sizeof(FPfield*), hipMemcpyDeviceToHost);
    hipMemcpy(&d_Ey_flat, &d_field->Ey_flat, sizeof(FPfield*), hipMemcpyDeviceToHost);
    hipMemcpy(&d_Ez_flat, &d_field->Ez_flat, sizeof(FPfield*), hipMemcpyDeviceToHost);

    hipMemcpy(d_Ex_flat, h_field->Ex_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyHostToDevice);
    hipMemcpy(d_Ey_flat, h_field->Ey_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyHostToDevice);
    hipMemcpy(d_Ez_flat, h_field->Ez_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyHostToDevice);

    FPfield* d_Bxn_flat, * d_Byn_flat, * d_Bzn_flat;
    hipMemcpy(&d_Bxn_flat, &d_field->Bxn_flat, sizeof(FPfield*), hipMemcpyDeviceToHost);
    hipMemcpy(&d_Byn_flat, &d_field->Byn_flat, sizeof(FPfield*), hipMemcpyDeviceToHost);
    hipMemcpy(&d_Bzn_flat, &d_field->Bzn_flat, sizeof(FPfield*), hipMemcpyDeviceToHost);

    hipMemcpy(d_Bxn_flat, h_field->Bxn_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyHostToDevice);
    hipMemcpy(d_Byn_flat, h_field->Byn_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyHostToDevice);
    hipMemcpy(d_Bzn_flat, h_field->Bzn_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyHostToDevice);
}

#endif // GPU
