

#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include <random>
#include <string>

#define NUM_BINS 4096
#define TPB 32

__global__ void histogram_kernel(unsigned int *input, unsigned int *bins,
                                 unsigned int num_elements,
                                 unsigned int num_bins) {

//@@ Insert code below to compute histogram of input using shared memory and atomics

  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  atomicAdd(&bins[input[i]], 1u);
}

__global__ void convert_kernel(unsigned int *bins, unsigned int num_bins) {

//@@ Insert code below to clean up bins that saturate at 127

  const unsigned int bin = blockIdx.x * blockDim.x + threadIdx.x;
  if(bins[bin] > 127) bins[bin] = 127;
}

class Timer{
  std::chrono::time_point<std::chrono::high_resolution_clock> start;

public:
  Timer(){
    start = std::chrono::high_resolution_clock::now();
  }

  double get(){
    auto stop = std::chrono::high_resolution_clock::now();
    return std::chrono::duration<double>(stop - start).count();
  }
};

int main(int argc, char **argv) {
  hipFree(0);

  int inputLength;
  unsigned int *hostInput;
  unsigned int *hostBins;
  unsigned int *resultRef;
  unsigned int *deviceInput;
  unsigned int *deviceBins;

  //@@ Insert code below to read in inputLength from args
  if (argc > 1)
    inputLength = std::stoi(argv[1]);
  printf("The input length is %d\n", inputLength);
  
  //@@ Insert code below to allocate Host memory for input and output
  hostInput = (unsigned int*) malloc(sizeof(unsigned int) * inputLength);
  hostBins = (unsigned int*) malloc(sizeof(unsigned int) * NUM_BINS);
  
  //@@ Insert code below to initialize hostInput to random numbers whose values range from 0 to (NUM_BINS - 1)
  std::random_device rd;
  std::mt19937 gen(rd());
  std::uniform_int_distribution<unsigned int> dis(0, NUM_BINS-1);
  for(int i = 0; i < inputLength; i++){
    hostInput[i] = dis(gen);
  }

  //@@ Insert code below to create reference result in CPU
  Timer cpuTimer;
  resultRef = (unsigned int*) malloc(sizeof(unsigned int) * NUM_BINS);
  for(int i = 0; i < inputLength; i++){
    unsigned int input = hostInput[i];
    if (resultRef[input] < 127)
      resultRef[input]++;
  }
  printf("CPU time: %fs\n", cpuTimer.get());

  Timer gpuTimer;
  //@@ Insert code below to allocate GPU memory here
  hipMalloc(&deviceInput, sizeof(unsigned int) * inputLength);
  
  //@@ Insert code to Copy memory to the GPU here
  hipMemcpy(deviceInput, hostInput, sizeof(unsigned int) * inputLength, hipMemcpyHostToDevice);
  
  //@@ Insert code to initialize GPU results
  hipMalloc(&deviceBins, sizeof(unsigned int) * NUM_BINS);


  //@@ Initialize the grid and block dimensions here
  const dim3 blockSizeHist(TPB);
  const dim3 gridSizeHist((inputLength + TPB - 1) / TPB);

  //@@ Launch the GPU Kernel here
  Timer histKernelTimer;
  histogram_kernel<<<gridSizeHist,blockSizeHist>>>(deviceInput, deviceBins, inputLength, NUM_BINS);
  printf("Histogram kernel time: %fs\n", histKernelTimer.get());
  
  //@@ Initialize the second grid and block dimensions here
  const dim3 blockSizeConv(TPB);
  const dim3 gridSizeConv((NUM_BINS + TPB - 1) / TPB);
  
  //@@ Launch the second GPU Kernel here
  Timer convKernelTimer;
  convert_kernel<<<gridSizeConv,blockSizeConv>>>(deviceBins, NUM_BINS);
  printf("Convert kernel time: %fs\n", convKernelTimer.get());

  //@@ Copy the GPU memory back to the CPU here
  hipMemcpy(hostBins, deviceBins, sizeof(unsigned int) * NUM_BINS, hipMemcpyDeviceToHost);
  
  printf("GPU time: %fs\n", gpuTimer.get());

  //@@ Insert code below to compare the output with the reference
  printf("VERIFYING\n");
  bool isCorrect = true;
  for (int i = 0; i < NUM_BINS; i++){
    if(hostBins[i] != resultRef[i]){
      printf("%d: %d \t %d\n", i, hostBins[i], resultRef[i]);
      isCorrect = false;
      break;
    }
  }

  if (isCorrect)
    printf("Correct\n");
  else
    printf("CPU and GPU results do not match\n");


  //@@ Free the GPU memory here
  hipFree(deviceInput);
  hipFree(deviceBins);

  //@@ Free the CPU memory here
  free(hostInput);
  free(hostBins);

  return 0;
}

