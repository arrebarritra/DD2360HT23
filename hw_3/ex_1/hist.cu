

#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <chrono>
#include <ctime>  
#include <sstream>
#include <iomanip>
#include <random>
#include <string>

#define NUM_BINS 4096
#define TPB_HIST 1024
#define TPB_CONV 32

__global__ void histogram_kernel(unsigned int *input, unsigned int *bins,
                                 unsigned int num_elements,
                                 unsigned int num_bins) {

//@@ Insert code below to compute histogram of input using shared memory and atomics
  // input element
  const int el = blockIdx.x * blockDim.x + threadIdx.x;
  // shared bins
  __shared__ unsigned int s_bins[NUM_BINS];
  
  int bin_region_size = num_bins / blockDim.x;
  int bin_fill_start = bin_region_size * threadIdx.x;
  
  // collaboratively set shared bins to 0
  for (int i = bin_fill_start; i < bin_fill_start + bin_region_size; i++)
    s_bins[i] = 0;
  __syncthreads();

  if (el < num_elements){  
    atomicAdd(&s_bins[input[el]], 1u);
  }
  __syncthreads();

  // collaboratively add to bins between threads
  for (int i = bin_fill_start; i < bin_fill_start + bin_region_size; i++) {
    unsigned int bincount = s_bins[i];
    if (bincount > 0)
      atomicAdd(&bins[i], bincount);
  }
}

__global__ void convert_kernel(unsigned int *bins, unsigned int num_bins) {

//@@ Insert code below to clean up bins that saturate at 127

  const unsigned int bin = blockIdx.x * blockDim.x + threadIdx.x;
  if(bin >= num_bins) return;
  unsigned int count = bins[bin];
  bins[bin] = count > 127 ? 127 : count;
}

class Timer{
  std::chrono::time_point<std::chrono::high_resolution_clock> start;

public:
  Timer(){
    start = std::chrono::high_resolution_clock::now();
  }

  double get(){
    auto stop = std::chrono::high_resolution_clock::now();
    return std::chrono::duration<double>(stop - start).count();
  }
};

void saveres(unsigned int inputLength, unsigned int* hist){
  auto now = std::chrono::system_clock::now();
  auto in_time_t = std::chrono::system_clock::to_time_t(now);

  std::stringstream fnss;
  fnss << "res/hist-" << std::to_string(inputLength) << "-";
  fnss << std::put_time(std::localtime(&in_time_t), "%Y-%m-%d-%H-%M-%S");
  fnss << ".txt";

  std::ofstream f (fnss.str().c_str());
  for(int i = 0; i < NUM_BINS; i++){
    f << hist[i] << std::endl;
  }
  
  f.close();
  printf("Result saved to: %s\n", fnss.str().c_str());
}

int main(int argc, char **argv) {
  hipFree(0);

  int inputLength;
  unsigned int *hostInput;
  unsigned int *hostBins;
  unsigned int *resultRef;
  unsigned int *deviceInput;
  unsigned int *deviceBins;

  //@@ Insert code below to read in inputLength from args
  if (argc > 1)
    inputLength = std::stoi(argv[1]);
  else{   
    printf("Provide input length");
    return;
  }
  printf("The input length is %d\n", inputLength);
  
  //@@ Insert code below to allocate Host memory for input and output
  hostInput = (unsigned int*) malloc(sizeof(unsigned int) * inputLength);
  hostBins = (unsigned int*) malloc(sizeof(unsigned int) * NUM_BINS);
  
  //@@ Insert code below to initialize hostInput to random numbers whose values range from 0 to (NUM_BINS - 1)
  std::random_device rd;
  std::minstd_rand gen(rd());
  std::uniform_int_distribution<unsigned int> dis(0, NUM_BINS-1);
  for(int i = 0; i < inputLength; i++){
    hostInput[i] = dis(gen);
  }

  //@@ Insert code below to create reference result in CPU
  Timer cpuTimer;
  resultRef = (unsigned int*) calloc(NUM_BINS, sizeof(unsigned int));
  for(int i = 0; i < inputLength; i++){
    unsigned int input = hostInput[i];
    if (resultRef[input] < 127)
      resultRef[input]++;
  }
  printf("CPU time: %fs\n", cpuTimer.get());

  Timer gpuTimer;
  //@@ Insert code below to allocate GPU memory here
  hipMalloc(&deviceInput, sizeof(unsigned int) * inputLength);
  
  //@@ Insert code to Copy memory to the GPU here
  hipMemcpy(deviceInput, hostInput, sizeof(unsigned int) * inputLength, hipMemcpyHostToDevice);
  
  //@@ Insert code to initialize GPU results
  hipMalloc(&deviceBins, sizeof(unsigned int) * NUM_BINS);
  hipMemset(deviceBins, 0u, sizeof(unsigned int) * NUM_BINS);

  //@@ Initialize the grid and block dimensions here
  const dim3 blockSizeHist(TPB_HIST);
  const dim3 gridSizeHist((inputLength + TPB_HIST - 1) / TPB_HIST);

  //@@ Launch the GPU Kernel here
  histogram_kernel<<<gridSizeHist,blockSizeHist>>>(deviceInput, deviceBins, inputLength, NUM_BINS);
  
  //@@ Initialize the second grid and block dimensions here
  const dim3 blockSizeConv(TPB_CONV);
  const dim3 gridSizeConv((NUM_BINS + TPB_CONV - 1) / TPB_CONV);
  
  //@@ Launch the second GPU Kernel here
  convert_kernel<<<gridSizeConv,blockSizeConv>>>(deviceBins, NUM_BINS);

  //@@ Copy the GPU memory back to the CPU here
  hipMemcpy(hostBins, deviceBins, sizeof(unsigned int) * NUM_BINS, hipMemcpyDeviceToHost);
  printf("GPU time: %fs\n", gpuTimer.get());

  //@@ Insert code below to compare the output with the reference
  printf("VERIFYING\n");
  bool isCorrect = true;
  for (int i = 0; i < NUM_BINS; i++){
    if(resultRef[i] != hostBins[i]){
      printf("%d: %d \t %d\n", i, resultRef[i], hostBins[i]);
      isCorrect = false;
      break;
    }
  }

  if (isCorrect){
    printf("Correct\n");
    saveres(inputLength, hostBins);
  }
  else
    printf("CPU and GPU results do not match\n");


  //@@ Free the GPU memory here
  hipFree(deviceInput);
  hipFree(deviceBins);

  //@@ Free the CPU memory here
  free(hostInput);
  free(hostBins);
  free(resultRef);

  return 0;
}

